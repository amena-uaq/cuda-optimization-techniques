﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>

#include "Matrix.cuh"

int main()
{
	// Matriz con manejo manual
	auto InicioContadorMManu = std::chrono::high_resolution_clock::now();
	MatrixManual MManuA(10000, 10000);
	MatrixManual MManuB(10000, 10000);

	MManuA.LLenarConValor(3.0);
	MManuB.LLenarConValor(2.0);

	MManuA.Sumar(MManuB);
	auto FinContadorMManu = std::chrono::high_resolution_clock::now();

	auto DuracionMManu = std::chrono::duration_cast<std::chrono::milliseconds>(FinContadorMManu - InicioContadorMManu);
	std::cout << "Tiempo transcurrido Matriz manejada manualmente: " << DuracionMManu.count() << " milisegundos\n";

	std::cout << MManuA;
	std::cout << MManuB;

	// Matriz con manejo automatico
	auto InicioContadorMMane = std::chrono::high_resolution_clock::now();
	MatrixManejada MManeA(10000, 10000);
	MatrixManejada MManeB(10000, 10000);

	MManeA.LLenarConValor(3.0);
	MManeB.LLenarConValor(2.0);

	MManeA.Sumar(MManeB);
	auto FinContadorMMane = std::chrono::high_resolution_clock::now();

	auto DuracionMMane = std::chrono::duration_cast<std::chrono::milliseconds>(FinContadorMMane - InicioContadorMMane);
	std::cout << "\n\nTiempo transcurrido Matriz manejada automaticamente: " << DuracionMMane.count() << " milisegundos\n";

	std::cout << MManeA;
	std::cout << MManeB;

	// Matriz con manejo manual y memoria fijada en el host
	auto InicioContadorMFij = std::chrono::high_resolution_clock::now();
	MatrizFijada MFijA(10000, 10000);
	MatrizFijada MFijB(10000, 10000);

	MFijA.LLenarConValor(3.0);
	MFijB.LLenarConValor(2.0);

	MFijA.Sumar(MFijB);
	auto FinContadorMFij = std::chrono::high_resolution_clock::now();

	auto DuracionMFij = std::chrono::duration_cast<std::chrono::milliseconds>(FinContadorMFij - InicioContadorMFij);
	std::cout << "\n\nTiempo transcurrido Matriz fijada: " << DuracionMFij.count() << " milisegundos\n";

	std::cout << MFijA;
	std::cout << MFijB;

	// Matriz con manejo manual, memoria fijada en el host y memoria asincrona en el device
	auto InicioContadorMFijAsync = std::chrono::high_resolution_clock::now();
	MatrizFijada MFijAsyncA(10000, 10000);
	MatrizFijada MFijAsyncB(10000, 10000);

	MFijAsyncA.LLenarConValor(3.0);
	MFijAsyncB.LLenarConValor(2.0);

	MFijAsyncA.Sumar(MFijAsyncB);
	auto FinContadorMFijAsync = std::chrono::high_resolution_clock::now();

	auto DuracionMFijAsync = std::chrono::duration_cast<std::chrono::milliseconds>(FinContadorMFijAsync - InicioContadorMFijAsync);
	std::cout << "\n\nTiempo transcurrido Matriz fijada: " << DuracionMFijAsync.count() << " milisegundos\n";

	std::cout << MFijAsyncA;
	std::cout << MFijAsyncB;
	return 0;
}
